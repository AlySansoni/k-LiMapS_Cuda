#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "../utils/common.h"
//#include <gsl/gsl_blas.h>
//#include <gsl/gsl_linalg.h>
#include <thrust/device_vector.h>
#include <hipsolver.h>
#include <hip/hip_runtime_api.h>
#include "utilities.cuh"

typedef float realtype;

#define N 800
#define M 1280
#define K 32
#define SHAREDBLOCKSIZE 32

#define max(a,b)		((a) > (b) ? (a) : (b))
#define min(a,b)		((a) < (b) ? (a) : (b))

#define SEED time(NULL)

#define MAXITER 1000

__host__ void matrixDisplay (float *arr1, int row, int col){
 
    for (int i = 0; i < row; i++){
        for(int j = 0; j<col; j++)
            printf("%f ", arr1[i*col+j]);
    printf("\n");
    }
    return;
}

__global__ void rand_gen_gpu(float *dict, hiprandState *states, int nRows, int nCols) {

    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	
    if (y < nRows && x < nCols)
        hiprand_init(y*nCols+x, 0, 0, &states[y*nCols+x]);
        dict[y*nCols+x] = hiprand_normal_double(&states[y*nCols+x]);

}

__global__ void setup_kernel ( hiprandState * state, unsigned long seed, int n){

    int id = threadIdx.x+blockDim.x*blockIdx.x;
    if (id < n)
      hiprand_init (seed, id, 0, &state[id] );
}

__global__ void generate_array( hiprandState* globalState, float * result, int count ){

    int ind = threadIdx.x+blockDim.x*blockIdx.x;
    if (ind < count){
      float tmp = hiprand_uniform( &globalState[ind] );
      result[ind] = int(abs(tmp*(M-1)));
    }
}

__host__ float euclNorm(float *arr, int dim){

    float elem;
    float sum= 0.0;

    for (int i = 0; i<dim; i++){    
        elem= arr[i];
        sum+= elem*elem;      
    }

    return (float)(sqrt(sum)); 
 }

__global__ void transposeSmem(float *in, float *out, int nrows, int ncols) {

    extern __shared__ float tile[];

    unsigned int row = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int col = blockDim.x * blockIdx.x + threadIdx.x;
 
    if (row < nrows && col < ncols)
        tile[threadIdx.y*blockDim.y+threadIdx.x] = in[row*ncols+col];
    __syncthreads();

    // transposed block offset
    int y = blockIdx.x * blockDim.x + threadIdx.y;
    int x = blockIdx.y * blockDim.y + threadIdx.x;

    // switched controls
    if (y < ncols && x < nrows)
        out[y*nrows + x] = tile[threadIdx.x*blockDim.x+threadIdx.y];

}

    // Kernel for matrix product using dynamic SMEM
__global__ void matProdSMEMdynamic(float* A, float* B, float* C, int row1, int col1, int col2, const uint SMEMsize) {
	// indexes
	uint row = blockIdx.y * blockDim.y + threadIdx.y;
	uint col = blockIdx.x * blockDim.x + threadIdx.x;

	// dynamic shared memory (inside or outside kernel)
	extern __shared__ float smem[];

	// Var As is manually set at beginning of shared
	float *As = smem;
	// Var Bs is manually set at the end of As
	float *Bs = &smem[SMEMsize];

	// loop over blocks from block row of matrix A
	// and block column of matrix B
	float sum = 0.0;
	uint numBlocks = (col1 + blockDim.x - 1) / blockDim.x;
	for (uint m = 0; m < numBlocks; m++) {

		// copy block from matrix to shared memory
		uint c = m * blockDim.x + threadIdx.x;
		uint r = m * blockDim.y + threadIdx.y;
		As[threadIdx.y * blockDim.y + threadIdx.x] = A[row*col1+c];//A[IDX(row, c, col1)];
		Bs[threadIdx.y * blockDim.y + threadIdx.x] = B[r*col2+col];//B[IDX(r, col, col2)];

		__syncthreads();

        //tuning the last block
		uint Z = (m == numBlocks - 1 ? col1 - m * blockDim.x : blockDim.x);

		// compute this part of row-column product
		for (int z = 0; z < Z; z++)
			sum += As[threadIdx.y * blockDim.x + z] * Bs[z * blockDim.y + threadIdx.x];

		__syncthreads();
	}

	// store computed element in matrix C
	if (row < row1 && col < col2)
		C[row * col2 + col] = sum;
}


__global__ void elemWise_mult(float *A, float *B, float *C, int numElements) {
	
    int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements)
		C[i] = A[i] * B[i];
}

__global__ void abs_array (float *arr, int dim){
    
    int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < dim){
        if(arr[i]<0.0)
            arr[i] = -arr[i];
    }	

    return;
    
}

__global__ void copy_arr (float *src, float*dest ,int dim){

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < dim)
        dest[i] = src[i];

}

__global__ void matrixDiff(float *A, float *B, float *C, int dim) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < dim)
	    C[i] = A[i] - B[i];
}

__global__ void arr_preProc(float *A, int dim){

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < dim)
        A[i] = exp(-A[i]);

}

__global__ void subMatrix(float *A, float*B, int *index, int nRows, int nCols){

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if (idx < nCols & idy < nRows){
        A[idy * nCols + idx] = B[index[idy]*nCols+idx];
    }


}
    
__global__ void copy_matrix(float *src, float *dest, int nRows, int nCols){
    
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    int id_elem= idy * nCols + idx;

    if (idy < nRows & idx < nCols)
        dest[id_elem] = src[id_elem];
}


__global__ void array_initialize(float *tmp_lambaMat, float lambda, int dim){

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < dim){
        tmp_lambaMat[i]=lambda;
    }
}

void moore_penrose_pinv(float* src, float *dst, int dim1, int dim2){
    
    dim3 blockShared(SHAREDBLOCKSIZE, SHAREDBLOCKSIZE);
    dim3 gridShared;
	uint SMEMsize = SHAREDBLOCKSIZE *SHAREDBLOCKSIZE;
	uint SMEMbyte = 2 * SMEMsize * sizeof(float);


    const realtype rcond = 1E-15;

    unsigned int n = dim1;
    unsigned int m = dim2;
    float *V, *Sigma_pinv, *U;
    float *tmp_U;
    float *_tmp_mat;
    float *s;
    int i;
    realtype x, cutoff;

    bool was_swapped = false;


    int blockSize = 32;
    dim3 block(blockSize, blockSize);
	dim3 grid1((m + block.x - 1) / block.x, (n + block.y - 1) / block.y);
    dim3 grid2((n + block.x - 1) / block.x, (m + block.y - 1) / block.y);

    if (m > n) {
		/* libgsl SVD caNonly handle the case M<= N- transpose matrix */
		was_swapped = true;
        CHECK(hipMallocManaged(&_tmp_mat, m*n*sizeof(float)));
        gridShared.y = (n + blockShared.y - 1) / blockShared.y;
        gridShared.x = (m+blockShared.x - 1) / blockShared.x;
        transposeSmem<<<gridShared, blockShared, SMEMbyte>>>(src, _tmp_mat, n, m);
        CHECK(hipDeviceSynchronize());

		copy_matrix<<<grid2, block>>>(_tmp_mat,src,m,n);
        CHECK(hipDeviceSynchronize());
		i = m;
		m = n;
		n = i;
	}

    if (was_swapped)
        CHECK(hipFree(_tmp_mat));

    /* do SVD, gsl version */
    /*CHECK(hipMallocManaged(&V,m*m*sizeof(float)));
    CHECK(hipMallocManaged(&s,m*sizeof(float)));

    gsl_matrix *tmp_src;
    gsl_matrix *tmp_V;
    gsl_vector *tmp_s;
    gsl_vector *_tmp_vec;
    tmp_src = gsl_matrix_alloc(n,m);
    for ( int row = 0; row < n; row++ ) { 
        for ( int col = 0; col < m; col++ ) { 
            gsl_matrix_set(tmp_src, row, col, src[row*m+col]);
        }
    }
    tmp_V = gsl_matrix_alloc(m, m);
	tmp_s = gsl_vector_alloc(m);
	_tmp_vec = gsl_vector_alloc(m);
	gsl_linalg_SV_decomp(tmp_src, tmp_V, tmp_s, _tmp_vec);
	gsl_vector_free(_tmp_vec);

    
    for ( int row = 0; row < n; row++ ) { 
        for ( int col = 0; col < m; col++ ) { 
            src[row*n+col] = gsl_matrix_get(tmp_src, row, col);
        }
    }
    gsl_matrix_free(tmp_src);

    for ( int row = 0; row < m; row++ ) { 
        for ( int col = 0; col < m; col++ ) { 
            V[row*m+col] = gsl_matrix_get(tmp_V, row, col);
        }
        s[row] = gsl_vector_get(tmp_s, row);
    }
    gsl_matrix_free(tmp_V);
    gsl_vector_free(tmp_s);

    // libgsl SVD yields "thin" SVD - pad to full matrix by adding zeros
    CHECK(hipMallocManaged(&U,n*n*sizeof(float)));
    CHECK(hipMemset(U,0,n*n*sizeof(float)));

    for(int i = 0; i<n; i++){
        for(int j=0; j<m; j++){
            U[i*n+j]=src[i*n+j];
        }
    }
    //dim3 grid3S((m + blockS.x - 1) / blockS.x, (m + blockS.y - 1) / blockS.y);
    //copy_matrix<<<grid3,block>>>(src,U,n,m);
    //CHECK(hipDeviceSynchronize());
    //printf("this is U:\n");
    //matrixDisplay(U,n,n);
	*/

     /* do SVD, cuSolver version */
    CHECK(hipMallocManaged(&V,m*m*sizeof(float)));
    CHECK(hipMallocManaged(&s,m*sizeof(float)));
    CHECK(hipMallocManaged(&tmp_U,n*m*sizeof(float)));
    CHECK(hipMallocManaged(&U,n*n*sizeof(float)));

    int work_size = 0;
    int *devInfo;          
    CHECK(hipMallocManaged(&devInfo,sizeof(int)));
    const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvectors.
    const int econ = 1 ; 

    hipsolverHandle_t solver_handle;
    hipsolverGesvdjInfo_t gesvdj_params;
    hipsolverDnCreate(&solver_handle);
    hipsolverDnCreateGesvdjInfo(&gesvdj_params);
    // --- CUDA SVD initialization
    cusolveSafeCall(hipsolverDnSgesvdj_bufferSize(solver_handle, jobz,econ,n, m, src, n,s,tmp_U,n,V,m,&work_size, gesvdj_params));

    float *work;   
    CHECK(hipMallocManaged(&work, work_size * sizeof(float)));
    // --- CUDA SVD execution
    cusolveSafeCall(hipsolverDnSgesvdj(solver_handle, jobz, econ , n, m, src, n, s, tmp_U,n , V, m, work, work_size, devInfo, gesvdj_params));
    CHECK(hipDeviceSynchronize());
  
    for(int i = 0; i<n; i++){
        for(int j=0; j<m; j++){
            U[i*n+j]=tmp_U[i*m+j];
        }
    }

    CHECK(hipFree(devInfo));
    CHECK(hipFree(work));
    CHECK(hipFree(tmp_U));
    cusolveSafeCall(hipsolverDnDestroy(solver_handle));
    cusolveSafeCall(hipsolverDnDestroyGesvdjInfo(gesvdj_params));


    /* compute Σ⁻¹ */
    CHECK(hipMallocManaged(&Sigma_pinv, m*n*sizeof(float)));
    CHECK(hipMemset(Sigma_pinv, 0.0, m*n*sizeof(float)));
    float *max_s = thrust::max_element(thrust::device, s, s + m);
    float tmp_max = *max_s;
	cutoff = rcond * tmp_max;

	for (i = 0; i < m; ++i) {
		if (s[i] > cutoff) {
			x = 1. / s[i];
		}
		else {
			x = 0.;
		}
        Sigma_pinv[i*n+i] = x;
	}

	/* two dot products to obtain pseudoinverse */
    CHECK(hipMallocManaged(&_tmp_mat,m*n*sizeof(float)));
    gridShared.y = (m + blockShared.y - 1) / blockShared.y;
    gridShared.x = (n+blockShared.x - 1) / blockShared.x;
    matProdSMEMdynamic<<<gridShared, blockShared, SMEMbyte>>>(V,Sigma_pinv,_tmp_mat,m,m,n, SMEMsize);
    CHECK(hipDeviceSynchronize());
    
	if (was_swapped) {
		transposeSmem<<<gridShared,blockShared, SMEMbyte>>>(_tmp_mat, src, m,n);
        CHECK(hipDeviceSynchronize());
        gridShared.y = (n + blockShared.y - 1) / blockShared.y;
        gridShared.x = (m+blockShared.x - 1) / blockShared.x;
        matProdSMEMdynamic<<<gridShared, blockShared, SMEMbyte>>>(U,src,dst,n,n,m, SMEMsize);
        CHECK(hipDeviceSynchronize());
	}
	else {
        float *tmp_U;
        CHECK(hipMallocManaged(&tmp_U, n*n*sizeof(float)));;

        gridShared.y = (n + blockShared.y - 1) / blockShared.y;
        gridShared.x = (n+blockShared.x - 1) / blockShared.x;
        transposeSmem<<<gridShared,blockShared, SMEMbyte>>>(U, tmp_U, n,n);
        CHECK(hipDeviceSynchronize());
        gridShared.y = (m + blockShared.y - 1) / blockShared.y;
        gridShared.x = (n+blockShared.x - 1) / blockShared.x;
        matProdSMEMdynamic<<<gridShared, blockShared, SMEMbyte>>>(_tmp_mat,tmp_U,dst,m,n,n, SMEMsize);

        CHECK(hipDeviceSynchronize());
        CHECK(hipFree(tmp_U));
	}

    CHECK(hipFree(_tmp_mat));
    CHECK(hipFree(U));
    CHECK(hipFree(Sigma_pinv));
    CHECK(hipFree(s));
    CHECK(hipFree(V));

}

void createDict_CPU(int n, int m, int k, float *D, float *Dinv, float *s) {

    dim3 blockShared(SHAREDBLOCKSIZE, SHAREDBLOCKSIZE);
    dim3 gridShared;
	uint SMEMsize = SHAREDBLOCKSIZE *SHAREDBLOCKSIZE;
	uint SMEMbyte = 2 * SMEMsize * sizeof(float);

    int blockSize2 = 1024;
    int block2(blockSize2);

    int blockSize = 32;

    dim3 block(blockSize, blockSize);

    srand(SEED);
   
    float *true_alpha;

    ulong mSize = m*sizeof(float);
    ulong nSize = n*sizeof(float);

    CHECK(hipMallocManaged(&true_alpha,mSize));
    CHECK(hipMemset(true_alpha, 0.0, mSize));

    for( int j = 0; j < k; j++){
            true_alpha[j] = (rand()%(n+1))-(float)(n/2);
    }
 
    float *tmp_perm_index;
    CHECK(hipMallocManaged(&tmp_perm_index,mSize));
    hiprandState *devStates2;
	CHECK(hipMalloc((void **) &devStates2, m* sizeof(hiprandState)));

    setup_kernel <<< (m+blockSize2-1)/blockSize2, block2 >>> ( devStates2, time(NULL), m );
    generate_array <<< (m+blockSize2-1)/blockSize, block2>>> ( devStates2, tmp_perm_index, m );
    CHECK(hipDeviceSynchronize());

    thrust::sort_by_key(thrust::device, tmp_perm_index, tmp_perm_index + m, true_alpha);
    
    //create random dictionary
    hiprandState *devStates;
	CHECK(hipMalloc((void **) &devStates, n* m* sizeof(hiprandState)));

    dim3 grid1((m + block.x - 1) / block.x, (n + block.y - 1) / block.y);
  
    rand_gen_gpu<<<grid1, block>>>(D, devStates, n, m);
    CHECK(hipDeviceSynchronize());

    float *norm_support;
    float *D_transp; 

    CHECK(hipMallocManaged(&norm_support,nSize));
    CHECK(hipMallocManaged(&D_transp,mSize*nSize));
   
    gridShared.y = (n + blockShared.y - 1) / blockShared.y;
    gridShared.x = (m+blockShared.x - 1) / blockShared.x;
    transposeSmem<<<gridShared,blockShared, SMEMbyte>>>(D,D_transp, n, m);
    CHECK(hipDeviceSynchronize());
    
    float norm;
   
    for(int i = 0; i < m; i++){
        for(int z = 0; z<n; z++){
            norm_support[z]=D_transp[i*n+z];
        }
        norm = euclNorm(norm_support,n);
        for(int j = 0; j < n; j ++){
            D_transp[i*n+j] = D_transp[i*n+j]/norm;
        }

    }

    dim3 grid2((n + block.x - 1) / block.x, (m + block.y - 1) / block.y);

    gridShared.y = (n + blockShared.y - 1) / blockShared.y;
    gridShared.x = (m+blockShared.x - 1) / blockShared.x;
    transposeSmem<<<gridShared,blockShared, SMEMbyte>>>(D_transp,D,m,n);
	CHECK(hipDeviceSynchronize());
    CHECK(hipFree(norm_support));
 
    float *mat_D;
    CHECK(hipMallocManaged(&mat_D,n*m));
    copy_matrix<<<grid1,block>>>(D,mat_D,n,m);
    CHECK(hipDeviceSynchronize());

    moore_penrose_pinv(mat_D, Dinv, n, m);

    CHECK(hipFree(mat_D));

    gridShared.y = (n + blockShared.y - 1) / blockShared.y;
    gridShared.x = (1+blockShared.x - 1) / blockShared.x;
    matProdSMEMdynamic<<<gridShared, blockShared, SMEMbyte>>>(D,true_alpha,s, n,m,1, SMEMsize);
    CHECK(hipDeviceSynchronize());

    CHECK(hipFree(D_transp));
    CHECK(hipFree(true_alpha));
 
    return;
}


void k_limaps(int n, int m, int k, float *s, float *D, float *Dinv, float *alpha){
   
	dim3 blockShared(SHAREDBLOCKSIZE, SHAREDBLOCKSIZE);
    dim3 gridShared;
	uint SMEMsize = SHAREDBLOCKSIZE *SHAREDBLOCKSIZE;
	uint SMEMbyte = 2 * SMEMsize * sizeof(float);

    uint blockSize = 32;
    dim3 block(blockSize, blockSize);
	uint blockSize2 = 1024;
    dim3 block2(blockSize2);

    ulong mSize = m*sizeof(float);
    ulong nSize = n*sizeof(float);
    ulong kSize = k*sizeof(float);

    //Initialization
    dim3 grid1((m+blockSize2-1)/blockSize2);

    gridShared.y = (m + blockShared.y - 1) / blockShared.y;
    gridShared.x = (1+blockShared.x - 1) / blockShared.x;
    matProdSMEMdynamic<<<gridShared, blockShared, SMEMbyte>>>(Dinv, s, alpha, m,n,1, SMEMsize);
    CHECK(hipDeviceSynchronize());

    //alpha = DINV*s;

    //I do the alpha transpose to make things easier, then transpose again
    float *t_alpha;
    CHECK(hipMallocManaged(&t_alpha, mSize));
    

    copy_arr<<<grid1, block2>>>(alpha, t_alpha, m);
    CHECK(hipDeviceSynchronize());
   
    abs_array<<<grid1, block2>>>(t_alpha, m);
    CHECK(hipDeviceSynchronize());
   

    thrust::sort(t_alpha, t_alpha + M);
    
    float lambda = 1/t_alpha[(m-1)-k];

    float epsilon=1E-5; //stopping criteria
   
    float *alphaold;
    float *beta;
    float *tmp_d_beta; 
    float *tmp_dinv_dBetaS;
    float *tmp_lambaMat;

    CHECK(hipMallocManaged(&alphaold, mSize));
    CHECK(hipMallocManaged(&beta, mSize));
    CHECK(hipMallocManaged(&tmp_d_beta, nSize));
    CHECK(hipMallocManaged(&tmp_dinv_dBetaS, mSize));
    CHECK(hipMallocManaged(&tmp_lambaMat, mSize));    
    
    dim3 grid2((n+blockSize2-1)/blockSize2);

    // CORE
    for(int extLoop = 0; extLoop < MAXITER; extLoop++){

        array_initialize<<<grid1,block2>>>(tmp_lambaMat,lambda, m);
        array_initialize<<<grid1,block2>>>(beta,float(1.0),m);
        CHECK(hipDeviceSynchronize());
        
        
        copy_arr<<<grid1, block2>>>(alpha, alphaold, m);
        CHECK(hipDeviceSynchronize());
        
        // apply sparsity constraction mapping: increase sparsity

        abs_array<<<grid1, block2>>>(alpha,m);
        CHECK(hipDeviceSynchronize());

        elemWise_mult<<<grid1, block2>>>(tmp_lambaMat,alpha, alpha, m); 
        CHECK(hipDeviceSynchronize());

        arr_preProc<<<grid1, block2>>>(alpha, m);
        CHECK(hipDeviceSynchronize()); 
      
        matrixDiff<<<grid1, block2>>>(beta,alpha,beta, m);
        CHECK(hipDeviceSynchronize());
       
        elemWise_mult<<<grid1, block2>>>(alphaold,beta, beta, m);
        CHECK(hipDeviceSynchronize());
        
       
        // apply the orthogonal projection alpha = beta-DINV*(D*beta-s); 
     
        gridShared.y = (n + blockShared.y - 1) / blockShared.y;
        gridShared.x = (1+blockShared.x - 1) / blockShared.x;
        matProdSMEMdynamic<<<gridShared, blockShared, SMEMbyte>>>(D, beta, tmp_d_beta, n,m,1, SMEMsize);
    
        CHECK(hipDeviceSynchronize());
        
        matrixDiff<<<grid2, block2>>>(tmp_d_beta,s, tmp_d_beta, n);
        CHECK(hipDeviceSynchronize());
       
        gridShared.y = (m + blockShared.y - 1) / blockShared.y;
        gridShared.x = (1+blockShared.x - 1) / blockShared.x;
        matProdSMEMdynamic<<<gridShared, blockShared, SMEMbyte>>>(Dinv, tmp_d_beta, tmp_dinv_dBetaS, m,n,1, SMEMsize);
    
        CHECK(hipDeviceSynchronize());
       
        matrixDiff<<<grid1, block2>>>(beta, tmp_dinv_dBetaS, alpha, m);
        CHECK(hipDeviceSynchronize());
        
        // update the lambda coefficient
        copy_arr<<<grid1, block2>>>(alpha,t_alpha,m);

        abs_array<<<grid1, block2>>>(t_alpha,m);

        CHECK(hipDeviceSynchronize());

        thrust::sort(t_alpha, t_alpha + M);


        lambda = 1/t_alpha[(m-1)-k];
        
        // check the stopping criteria
        matrixDiff<<<grid1, block2>>>(alpha, alphaold, alphaold, m);
        CHECK(hipDeviceSynchronize());


        if (euclNorm(alphaold, m)<epsilon|| isnan(lambda)){
            printf("eucl norm: %f\n",euclNorm(alphaold,m));
            printf("Lambda: %f\n",lambda);
            printf("I'm exiting main core with break rule\n");
            break;
        }
    }

    CHECK(hipFree(tmp_d_beta));
    CHECK(hipFree(tmp_dinv_dBetaS));
    CHECK(hipFree(t_alpha));
    CHECK(hipFree(tmp_lambaMat));

    // FINAL REFINEMENTS FOR SOLUTION

    //I'll use beta again just to not allocating another useless variable
    int *idx_array;
    CHECK(hipMallocManaged(&idx_array, k*sizeof(int)));
    int count = 0;

    copy_arr<<<grid1, block2>>>(alpha, beta, m);

    abs_array<<<grid1, block2>>>(beta,m);
    CHECK(hipDeviceSynchronize());

    float *sel_alpha;
    CHECK(hipMallocManaged(&sel_alpha, kSize));

    for (int i=0; i<m; i++){
        if(beta[i] <= 1/lambda){
            alpha[i] = 0;
        }
        else{
            idx_array[count] = i;
            sel_alpha[count] = alpha[i];
            count++;
        }
    }

    float *D1;
    float *D1_transp;
    float *D_transp;

    CHECK(hipMallocManaged(&D1, nSize*kSize));
    CHECK(hipMallocManaged(&D1_transp, kSize*nSize));
    CHECK(hipMallocManaged(&D_transp, mSize*mSize));

    gridShared.y = (n + blockShared.y - 1) / blockShared.y;
    gridShared.x = (m+blockShared.x - 1) / blockShared.x;
    transposeSmem<<<gridShared, blockShared, SMEMbyte>>>(D,D_transp,n,m); 
    CHECK(hipDeviceSynchronize());
    
    dim3 grid4((n + block.x - 1) / block.x, (k + block.y - 1) / block.y);
    subMatrix<<<grid4, block>>>(D1_transp,D_transp, idx_array,k,n);
    CHECK(hipDeviceSynchronize());
  
    gridShared.y = (k + blockShared.y - 1) / blockShared.y;
    gridShared.x = (n+blockShared.x - 1) / blockShared.x;
    transposeSmem<<<gridShared, blockShared, SMEMbyte>>>(D1_transp,D1, k,n);
    CHECK(hipDeviceSynchronize());
    
    float *tmp_d1_alpha_mul;
    float *D1_pinv;
    float *tmp_pinvD1_par;
    CHECK(hipMallocManaged(&tmp_d1_alpha_mul, nSize));
    CHECK(hipMallocManaged(&D1_pinv, kSize*nSize));
    CHECK(hipMallocManaged(&tmp_pinvD1_par, kSize));

    gridShared.y = (n + blockShared.y - 1) / blockShared.y;
    gridShared.x = (1+blockShared.x - 1) / blockShared.x;
    matProdSMEMdynamic<<<gridShared, blockShared, SMEMbyte>>>(D1, sel_alpha, tmp_d1_alpha_mul, n,k,1, SMEMsize);

    CHECK(hipDeviceSynchronize()); 
   
    matrixDiff<<<grid2, block2>>>(tmp_d1_alpha_mul, s,tmp_d1_alpha_mul,n);
    CHECK(hipDeviceSynchronize());


    float *mat_D1;
    CHECK(hipMallocManaged(&mat_D1,n*k));

    copy_matrix<<<grid4, block>>>(D1,mat_D1,n,k);
    CHECK(hipDeviceSynchronize());
    
    moore_penrose_pinv(mat_D1, D1_pinv, n, k);
    CHECK(hipDeviceSynchronize());
    
    CHECK(hipFree(mat_D1));
  
    dim3 grid6((k+blockSize2-1)/blockSize2);
      
    gridShared.y = (k + blockShared.y - 1) / blockShared.y;
    gridShared.x = (1+blockShared.x - 1) / blockShared.x;  
    matProdSMEMdynamic<<<gridShared, blockShared, SMEMbyte>>>(D1_pinv,tmp_d1_alpha_mul, tmp_pinvD1_par, k, n, 1, SMEMsize);  
    
    CHECK(hipDeviceSynchronize());
 
    matrixDiff<<<grid6, block2>>>(sel_alpha, tmp_pinvD1_par ,sel_alpha, k);   
    CHECK(hipDeviceSynchronize());


    for(int i = 0; i< k; i++){
         alpha[idx_array[i]] = sel_alpha[i];
    }

    CHECK(hipFree(idx_array));
    CHECK(hipFree(beta));
    CHECK(hipFree(alphaold));
    CHECK(hipFree(sel_alpha));
    CHECK(hipFree(D1));
    CHECK(hipFree(tmp_d1_alpha_mul));
    CHECK(hipFree(D1_pinv));
    CHECK(hipFree(tmp_pinvD1_par));
    CHECK(hipFree(D1_transp));
    CHECK(hipFree(D_transp));

    return;
}

int main(int argc, char *argv[]) {

    float *D, *Dinv, *s, *alpha;
    ulong nSize = N* sizeof(float);
    ulong mSize = M* sizeof(float);

    CHECK(hipMallocManaged(&D, nSize*mSize));
    CHECK(hipMallocManaged(&Dinv, nSize*mSize));
    CHECK(hipMallocManaged(&s, nSize));
    CHECK(hipMallocManaged(&alpha, mSize));

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(matProdSMEMdynamic), hipFuncCachePreferShared);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(transposeSmem), hipFuncCachePreferShared);


    hipEvent_t start, stop;
    float cuTime;

    hipEventCreate(&start);
    hipEventRecord(start,0);
   

    createDict_CPU(N,M,K,D,Dinv, s);
    k_limaps(N, M, K, s,D, Dinv, alpha);
    hipEventCreate(&stop);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cuTime, start,stop);
    printf("The resulting alpha is:\n");
    matrixDisplay(alpha, M,1);
    printf("\n\nTotal time in GPU: %f ms \n\n", cuTime);

    CHECK(hipFree(D));
    CHECK(hipFree(Dinv));
    CHECK(hipFree(s));
    CHECK(hipFree(alpha));

    hipError_t cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

    return 0;
	
}