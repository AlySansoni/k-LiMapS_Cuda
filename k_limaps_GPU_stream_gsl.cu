#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "utils/common.h"
#include <gsl/gsl_blas.h>
#include <gsl/gsl_linalg.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime_api.h>

typedef float realtype;

#define N 600
#define M 700
#define K 32
#define SHAREDBLOCKSIZE 16
#define NSTREAM 8

#define max(a,b)		((a) > (b) ? (a) : (b))
#define min(a,b)		((a) < (b) ? (a) : (b))

#define SEED time(NULL)

#define MAXITER 1000

__host__ void matrixDisplay (float *arr1, int row, int col){
 
    for (int i = 0; i < row; i++){
        for(int j = 0; j<col; j++)
            printf("%f ", arr1[i*col+j]);
    printf("\n");
    }
    return;
}

__global__ void rand_gen_gpu(float *dict, hiprandState *states, int nRows, int nCols) {

    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < nRows && x < nCols)
        hiprand_init(y*nCols+x, 0, 0, &states[y*nCols+x]);
        dict[y*nCols+x] = hiprand_normal_double(&states[y*nCols+x]);

}

__global__ void setup_kernel ( hiprandState * state, unsigned long seed, int n)
{
    int id = threadIdx.x+blockDim.x*blockIdx.x;
    if (id < n)
      hiprand_init (seed, id, 0, &state[id] );
}

__global__ void generate_array( hiprandState* globalState, float * result, int count )
{
    int ind = threadIdx.x+blockDim.x*blockIdx.x;
    if (ind < count){
      float tmp = hiprand_uniform( &globalState[ind] );
      result[ind] = int(abs(tmp*(M-1)));
    }
}

__host__ float euclNorm(float *arr, int dim){

    float elem;
    float sum= 0.0;

    for (int i = 0; i<dim; i++){    
        elem= arr[i];
        sum+= elem*elem;      
    }

    return (float)(sqrt(sum)); 
 }

__global__ void transposeSmem(float *in, float *out, int nrows, int ncols) {

    __shared__ float tile[SHAREDBLOCKSIZE][SHAREDBLOCKSIZE];

    unsigned int row = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int col = blockDim.x * blockIdx.x + threadIdx.x;
    // from global to shared memory
    if (row < nrows && col < ncols)
        tile[threadIdx.y][threadIdx.x] = in[row*ncols+col];
    // thread synchronization
    __syncthreads();

    // transposed block offset
    int y = blockIdx.x * blockDim.x + threadIdx.y;
    int x = blockIdx.y * blockDim.y + threadIdx.x;
    // switched controls
    if (y < ncols && x < nrows)
        out[y*nrows + x] = tile[threadIdx.x][threadIdx.y];

}

__global__ void matrixMultStream(float* A, float* B, float* C, int row1, int col1, int col2, uint offset) {

    int Row = offset+blockIdx.y * blockDim.y + threadIdx.y;
    int Col = offset+blockIdx.x * blockDim.x + threadIdx.x;
    // each thread computes an entry of the product matrix 
    if ((Row < row1) && (Col < col2)) {
        float val = 0;
        for (int z= 0; z< col1; z++)
            val += A[Row * col1 + z] * B[z* col2 + Col];
        C[Row * col2 + Col] = val;
    }
}

__global__ void elemWise_mult(float *A, float *B, float *C, int numElements, uint offset) {
	
    int i = offset+blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements)
		C[i] = A[i] * B[i];
}

__global__ void abs_array (float *arr, int dim, uint offset){
    
    int i = offset+blockDim.x * blockIdx.x + threadIdx.x;

	if (i < dim){
        if(arr[i]<0.0)
            arr[i] = -arr[i];
    }	

    return;
    
}

__global__ void copy_arr (float *src, float*dest ,int dim, uint offset){

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < dim)
        dest[i] = src[i];

}

__global__ void matrixDiff(float *A, float *B, float *C, int dim, uint offset) {

    int i = offset+blockDim.x * blockIdx.x + threadIdx.x;

    if(i < dim)
	    C[i] = A[i] - B[i];
}


__global__ void arr_preProc(float *A, int dim, uint offset){

    int i = offset+blockDim.x * blockIdx.x + threadIdx.x;

    if(i < dim)
        A[i] = exp(-A[i]);

}

__global__ void subMatrix(float *A, float*B, int *index, int nRows, int nCols){

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if (idx < nCols & idy < nRows){
        A[idy * nCols + idx] = B[index[idy]*nCols+idx];
    }


}
    
__global__ void copy_matrix(float *src, float *dest, int nRows, int nCols, uint offset){
    
    int idx = offset + blockDim.x * blockIdx.x + threadIdx.x;
    int idy = offset + blockDim.y * blockIdx.y + threadIdx.y;

    int id_elem= idy * nCols + idx;

    if (idy < nRows & idx < nCols)
        dest[id_elem] = src[id_elem];
}

__global__ void array_initialize(float *tmp_lambaMat, float lambda, int dim, uint offset){

    int i = offset+blockDim.x * blockIdx.x + threadIdx.x;
    if (i < dim){
        tmp_lambaMat[i]=lambda;
    }
}

void moore_penrose_pinv(float* src, float *dst, int dim1, int dim2){
    
    dim3 blockShared(SHAREDBLOCKSIZE, SHAREDBLOCKSIZE);
    dim3 gridShared;

    const realtype rcond = 1E-15;

    unsigned int n = dim1;
    unsigned int m = dim2;
    float *V, *Sigma_pinv, *U;
    float *_tmp_mat;
    float *s;
    int i;
    realtype x, cutoff;

    bool was_swapped = false;

    hipStream_t stream[NSTREAM];



    int blockSize = 32;
    dim3 block(blockSize, blockSize);
	dim3 grid1((m + block.x - 1) / block.x, (n + block.y - 1) / block.y);
    dim3 grid2((n + block.x - 1) / block.x, (m + block.y - 1) / block.y);
  
    for (int i = 0; i < NSTREAM; ++i)
        CHECK(hipStreamCreate(&stream[i]));
   
    int iElem =  ((m*n)%NSTREAM == 0) ? (m*n)/NSTREAM : (m*n)/ NSTREAM+1;
    dim3 grid1St ((iElem/m+ blockShared.x - 1) / blockShared.x, (iElem/n + blockShared.y - 1) / blockShared.y);
    dim3 grid2St ((iElem/n+ blockShared.x - 1) / blockShared.x, (iElem/m+ blockShared.y - 1) / blockShared.y);

    if (m > n) {
		/* libgsl SVD caNonly handle the case M<= N- transpose matrix */
		was_swapped = true;
        CHECK(hipMallocManaged(&_tmp_mat, m*n*sizeof(float)));
        gridShared.y = (n + blockShared.y - 1) / blockShared.y;
        gridShared.x = (m+blockShared.x - 1) / blockShared.x;

        transposeSmem<<<gridShared, blockShared>>>(src, _tmp_mat, n, m); 
        CHECK(hipDeviceSynchronize());
        
        for (int z=0; z< NSTREAM; z++){
            int ioffset = z*iElem;
            copy_matrix<<<grid2St, blockShared, 0, stream[z]>>>(_tmp_mat,src,m,n,ioffset);
        }    

        CHECK(hipDeviceSynchronize());

		i = m;
		m = n;
		n = i;
	}

    if (was_swapped)
        CHECK(hipFree(_tmp_mat));

    /* do SVD */
    CHECK(hipMallocManaged(&V,m*m*sizeof(float)));
    CHECK(hipMallocManaged(&s,m*sizeof(float)));

    gsl_matrix *tmp_src;
    gsl_matrix *tmp_V;
    gsl_vector *tmp_s;
    gsl_vector *_tmp_vec;
    tmp_src = gsl_matrix_alloc(n,m);
    for ( int row = 0; row < n; row++ ) { 
        for ( int col = 0; col < m; col++ ) { 
            gsl_matrix_set(tmp_src, row, col, src[row*m+col]);
        }
    }
    tmp_V = gsl_matrix_alloc(m, m);
	tmp_s = gsl_vector_alloc(m);
	_tmp_vec = gsl_vector_alloc(m);
	gsl_linalg_SV_decomp(tmp_src, tmp_V, tmp_s, _tmp_vec);
	gsl_vector_free(_tmp_vec);

    
    for ( int row = 0; row < n; row++ ) { 
        for ( int col = 0; col < m; col++ ) { 
            src[row*n+col] = gsl_matrix_get(tmp_src, row, col);
        }
    }
    gsl_matrix_free(tmp_src);

    for ( int row = 0; row < m; row++ ) { 
        for ( int col = 0; col < m; col++ ) { 
            V[row*m+col] = gsl_matrix_get(tmp_V, row, col);
        }
        s[row] = gsl_vector_get(tmp_s, row);
    }

    gsl_matrix_free(tmp_V);
    gsl_vector_free(tmp_s);
	
    /* compute Σ⁻¹ */
    CHECK(hipMallocManaged(&Sigma_pinv, m*n*sizeof(float)));
    CHECK(hipMemset(Sigma_pinv, 0.0, m*n*sizeof(float)));
    float *max_s = thrust::max_element(thrust::device, s, s + m);
    float tmp_max = *max_s;
	cutoff = rcond * tmp_max;

	for (i = 0; i < m; ++i) {
		if (s[i] > cutoff) {
			x = 1. / s[i];
		}
		else {
			x = 0.;
		}
        Sigma_pinv[i*n+i] = x;
	}

	/*libgsl SVD yields "thin" SVD - pad to full matrix by adding zeros */
    CHECK(hipMallocManaged(&U,n*n*sizeof(float)));
    CHECK(hipMemset(U,0,n*n*sizeof(float)));

    for(int i = 0; i<n; i++){
        for(int j=0; j<m; j++){
            U[i*n+j]=src[i*n+j];
        }
    }

	/* two dot products to obtain pseudoinverse */
    CHECK(hipMallocManaged(&_tmp_mat,m*n*sizeof(float)));

    for (int i=0; i< NSTREAM; i++){
        int ioffset = i*iElem;
        matrixMultStream<<<grid1St, blockShared, 0, stream[i]>>>(V,Sigma_pinv,_tmp_mat,m,m,n, ioffset);
    }

    CHECK(hipDeviceSynchronize());

	if (was_swapped) {
		transposeSmem<<<gridShared,blockShared>>>(_tmp_mat, src, m,n);
        CHECK(hipDeviceSynchronize());

        for (int i=0; i< NSTREAM; i++){
            int ioffset = i*iElem;
            matrixMultStream<<<grid2St, blockShared, 0, stream[i]>>>(U,src,dst,n,n,m, ioffset);
        }
        CHECK(hipDeviceSynchronize());
	}
	else {
        float *tmp_U;
        CHECK(hipMallocManaged(&tmp_U, n*n*sizeof(float)));

        gridShared.y = (n + blockShared.y - 1) / blockShared.y;
        gridShared.x = (n+blockShared.x - 1) / blockShared.x;
        transposeSmem<<<gridShared,blockShared>>>(U, tmp_U, n,n);
        CHECK(hipDeviceSynchronize());

        for (int i=0; i< NSTREAM; i++){
            int ioffset = i*iElem;
            matrixMultStream<<<grid1St, blockShared, 0, stream[i]>>>(_tmp_mat,tmp_U,dst,m,n,n, ioffset);
        }

        CHECK(hipDeviceSynchronize());
        CHECK(hipFree(tmp_U));
	}
    for (int i = 0; i < NSTREAM; ++i)
        CHECK(hipStreamDestroy(stream[i]));

    CHECK(hipFree(_tmp_mat));
    CHECK(hipFree(U));
    CHECK(hipFree(Sigma_pinv));
    CHECK(hipFree(s));
    CHECK(hipFree(V));

}

void createDict_CPU(int n, int m, int k, float *D, float *Dinv, float *s) {

    dim3 blockShared(SHAREDBLOCKSIZE, SHAREDBLOCKSIZE);
    dim3 gridShared;

    int blockSize = 32;

    dim3 block(blockSize, blockSize);

    hipStream_t stream[NSTREAM];

    for (int i = 0; i < NSTREAM; ++i)
        CHECK(hipStreamCreate(&stream[i]));

    srand(SEED);
   
    float *true_alpha;

    ulong mSize = m*sizeof(float);
    ulong nSize = n*sizeof(float);

    CHECK(hipMallocManaged(&true_alpha,mSize));
    CHECK(hipMemset(true_alpha, 0.0, mSize));

    for( int j = 0; j < k; j++){
            true_alpha[j] = (rand()%(n+1))-(float)(n/2);
    }
    
    float *tmp_perm_index;
    CHECK(hipMallocManaged(&tmp_perm_index,mSize));
    hiprandState *devStates2;
	CHECK(hipMalloc((void **) &devStates2, m* sizeof(hiprandState)));

    setup_kernel <<< (m+1023)/1024, 1024 >>> ( devStates2, time(NULL), m );
    generate_array <<< (m+1023)/1024, 1024>>> ( devStates2, tmp_perm_index, m );
    CHECK(hipDeviceSynchronize());

    thrust::sort_by_key(thrust::device, tmp_perm_index, tmp_perm_index + m, true_alpha);

    //create random dictionary
    hiprandState *devStates;
	CHECK(hipMalloc((void **) &devStates, n* m* sizeof(hiprandState)));

    dim3 grid1((m + block.x - 1) / block.x, (n + block.y - 1) / block.y);
  
    rand_gen_gpu<<<grid1, block>>>(D, devStates, n, m);
    CHECK(hipDeviceSynchronize());

    float *norm_support;
    float *D_transp; 

    CHECK(hipMallocManaged(&norm_support,nSize));
    CHECK(hipMallocManaged(&D_transp,mSize*nSize));
   
    gridShared.y = (n + blockShared.y - 1) / blockShared.y;
    gridShared.x = (m+blockShared.x - 1) / blockShared.x;
    transposeSmem<<<gridShared,blockShared>>>(D,D_transp, n, m);
    CHECK(hipDeviceSynchronize());
  
    float norm;
    for(int i = 0; i < m; i++){
        for(int z = 0; z<n; z++){
            norm_support[z]=D_transp[i*n+z];
        }
        norm = euclNorm(norm_support,n);
        for(int j = 0; j < n; j ++){
            D_transp[i*n+j] = D_transp[i*n+j]/norm;
        }

    }

    dim3 grid2((n + block.x - 1) / block.x, (m + block.y - 1) / block.y);
    gridShared.y = (n + blockShared.y - 1) / blockShared.y;
    gridShared.x = (m+blockShared.x - 1) / blockShared.x;
    transposeSmem<<<gridShared,blockShared>>>(D_transp,D,m,n);
	CHECK(hipDeviceSynchronize());
    CHECK(hipFree(norm_support));

    float *mat_D;
    CHECK(hipMallocManaged(&mat_D,n*m));

    int iElem0 =  ((n*m)%NSTREAM == 0) ? (n*m)/NSTREAM : (n*m)/ NSTREAM+1;
    dim3 grid0St ((iElem0/n+ blockShared.x - 1) / blockShared.x, (iElem0/m + blockShared.y - 1) / blockShared.y);

    for (int i=0; i< NSTREAM; i++){
        int ioffset = i*iElem0;
        copy_matrix<<<grid0St,blockShared,0,stream[i]>>>(D,mat_D,n,m,ioffset);

    }   

    CHECK(hipDeviceSynchronize());

    moore_penrose_pinv(mat_D, Dinv, n, m);
    
    CHECK(hipFree(mat_D));

   
    dim3 grid1St ((iElem0/n+ blockShared.x - 1) / blockShared.x, (iElem0 + blockShared.y - 1) / blockShared.y);

    for (int i=0; i< NSTREAM; i++){
        int ioffset = i*iElem0;
        matrixMultStream<<<grid1St, blockShared, 0, stream[i]>>>(D,true_alpha,s, n,m,1, ioffset);
    }

    CHECK(hipDeviceSynchronize());
    for (int i = 0; i < NSTREAM; ++i)
        CHECK(hipStreamDestroy(stream[i]));

    CHECK(hipFree(D_transp));
    CHECK(hipFree(true_alpha));
 
    return;
}


void k_limaps(int n, int m, int k, float *s, float *D, float *Dinv, float *alpha){
   
	dim3 blockShared(SHAREDBLOCKSIZE, SHAREDBLOCKSIZE);
    dim3 blockShared2(SHAREDBLOCKSIZE*SHAREDBLOCKSIZE);
    dim3 gridShared;

    uint blockSize = 32;
    dim3 block(blockSize, blockSize);
	uint blockSize2 = 1024;
    dim3 block2(1024);

    ulong mSize = m*sizeof(float);
    ulong nSize = n*sizeof(float);
    ulong kSize = k*sizeof(float);

    hipStream_t stream[NSTREAM];

    for (int i = 0; i < NSTREAM; ++i)
        CHECK(hipStreamCreate(&stream[i]));

    int iElem = (m%NSTREAM == 0) ? m/NSTREAM : m/ NSTREAM+1;
    int iElem2 =  (n%NSTREAM == 0) ? n/NSTREAM :n/NSTREAM+1;
    
    dim3 grid1St ((iElem + block2.x - 1) / block2.x);
    dim3 grid2St((iElem2+block2.x-1)/block2.x);

    //Initialization
    dim3 grid1((m+blockSize2-1)/blockSize2);

    gridShared.y = (m + blockShared.y - 1) / blockShared.y;
    gridShared.x = (1+blockShared.x - 1) / blockShared.x;
    
    int iElemM1 =  (m%NSTREAM == 0) ? m/NSTREAM : m/ NSTREAM+1;

    int iElemTmp = ((m*n)%NSTREAM == 0) ? (m*n)/NSTREAM : (m*n)/ NSTREAM+1;
    dim3 grid1MSt ((iElemTmp/m+ blockShared.x - 1) / blockShared.x, (iElemTmp + blockShared.y - 1) / blockShared.y);

    for (int i=0; i< NSTREAM; i++){
        int ioffset = i*iElemTmp;
        matrixMultStream<<<grid1MSt, blockShared, 0, stream[i]>>>(Dinv, s, alpha, m,n,1,ioffset);
    }
    CHECK(hipDeviceSynchronize());

    //I do the alpha transpose to make things easier, then transpose again
    float *t_alpha;
    CHECK(hipMallocManaged(&t_alpha, mSize));
    
    for (int i = 0; i < NSTREAM; ++i) {
        int ioffset = i * iElem;
        copy_arr<<<grid1St, block2, 0, stream[i]>>>(&alpha[ioffset], &t_alpha[ioffset], m, ioffset);
        CHECK(hipStreamSynchronize(stream[i]));
        abs_array<<<grid1St, block2, 0,stream[i]>>>(t_alpha, m, ioffset);

    }

    thrust::sort(t_alpha, t_alpha + M);
    
    float lambda = 1/t_alpha[(m-1)-k];

    float epsilon=1E-5; //stopping criteria
   
    float *alphaold;
    float *beta;
    float *tmp_d_beta; 
    float *tmp_dinv_dBetaS;
    float *tmp_lambaMat;

    CHECK(hipMallocManaged(&alphaold, mSize));
    CHECK(hipMallocManaged(&beta, mSize));
    CHECK(hipMallocManaged(&tmp_d_beta, nSize));
    CHECK(hipMallocManaged(&tmp_dinv_dBetaS, mSize));
    CHECK(hipMallocManaged(&tmp_lambaMat, mSize));
    
    dim3 grid2((n+blockSize2-1)/blockSize2);
    
    int iElemM2 =  (n%NSTREAM == 0) ? n/NSTREAM : n/ NSTREAM+1;

    dim3 grid2MSt ((iElemM2/n+ blockShared.x - 1) / blockShared.x, (iElemM2 + blockShared.y - 1) / blockShared.y);

    // CORE
    for(int extLoop = 0; extLoop < MAXITER; extLoop++){

        for(int i=0; i<NSTREAM; ++i){
            int ioffset = i * iElem;
            copy_arr<<<grid1St, block2, 0, stream[i]>>>(&alpha[ioffset], &alphaold[ioffset], m, ioffset);
            abs_array<<<grid1St, block2, 0, stream[i]>>>(alpha,m, ioffset);
        }
        CHECK(hipDeviceSynchronize());

        for (int i = 0; i < NSTREAM; ++i) {
            int ioffset = i * iElem;
            array_initialize<<<grid1St,block2,0,stream[i]>>>(tmp_lambaMat,lambda, m, ioffset);
            array_initialize<<<grid1St,block2,0,stream[i]>>>(beta,float(1.0),m,ioffset);
        }
            
        CHECK(hipDeviceSynchronize());
        for (int i = 0; i < NSTREAM; ++i) {
            int ioffset = i * iElem;
            elemWise_mult<<<grid1St, block2, 0, stream[i]>>>(tmp_lambaMat,alpha, alpha, m, ioffset); 
        }
        CHECK(hipDeviceSynchronize());
        for (int i = 0; i < NSTREAM; ++i) {
            int ioffset = i * iElem;
            arr_preProc<<<grid1St, block2, 0, stream[i]>>>(alpha, m, ioffset);
        }   
        CHECK(hipDeviceSynchronize()); 
      
        for (int i = 0; i < NSTREAM; ++i) {
            int ioffset = i * iElem;
            matrixDiff<<<grid1St, block2, 0, stream[i]>>>(beta,alpha,beta, m, ioffset);
        }
        CHECK(hipDeviceSynchronize());
    
        for (int i = 0; i < NSTREAM; ++i) {
            int ioffset = i * iElem;
            elemWise_mult<<<grid1St, block2, 0, stream[i]>>>(alphaold,beta, beta, m, ioffset);
        }
        

        CHECK(hipDeviceSynchronize());
    
        for (int i=0; i< NSTREAM; i++){
            int ioffset = i*iElemM2;
            matrixMultStream<<<grid2MSt, blockShared, 0, stream[i]>>>(D, beta, tmp_d_beta, n,m,1,ioffset);
        }

        CHECK(hipDeviceSynchronize());
     
        for (int i = 0; i < NSTREAM; ++i) {
            int ioffset = i * iElem2;
            matrixDiff<<<grid2St, block2, 0, stream[i]>>>(tmp_d_beta,s, tmp_d_beta, n, ioffset);
        }
        CHECK(hipDeviceSynchronize());
        
        for (int i = 0; i < NSTREAM; ++i) {
            int ioffset = i * iElemM1;
            matrixMultStream<<<grid1St, blockShared,0, stream[i]>>>(Dinv, tmp_d_beta, tmp_dinv_dBetaS, m,n,1,ioffset);
        }

        CHECK(hipDeviceSynchronize());
        
        for (int i = 0; i < NSTREAM; ++i) {
            int ioffset = i * iElem;
            matrixDiff<<<grid1St, block2, 0, stream[i]>>>(beta, tmp_dinv_dBetaS, alpha, m, ioffset);
        
            // update the lambda coefficient
            copy_arr<<<grid1St, block2, 0, stream[i]>>>(&alpha[ioffset],&t_alpha[ioffset],m, ioffset);

            abs_array<<<grid1St, block2, 0, stream[i]>>>(t_alpha,m,ioffset);
        }
        

        CHECK(hipDeviceSynchronize());

        thrust::sort(t_alpha, t_alpha + M);


        lambda = 1/t_alpha[(m-1)-k];
      
        for (int i = 0; i < NSTREAM; ++i) {
            int ioffset = i * iElem;
            matrixDiff<<<grid1St, block2, 0,stream[i]>>>(alpha, alphaold, alphaold, m, ioffset);
        }
        
        CHECK(hipDeviceSynchronize());
        // check the stopping criteria
        if (euclNorm(alphaold, m)<epsilon|| isnan(lambda)){
            printf("eucl norm: %f\n",euclNorm(alphaold,m));
            printf("Lambda: %f\n",lambda);
            printf("I'm exiting main core with break rule\n");
            break;
        }
    }

    CHECK(hipFree(tmp_d_beta));
    CHECK(hipFree(tmp_dinv_dBetaS));
    CHECK(hipFree(t_alpha));
    CHECK(hipFree(tmp_lambaMat));
 
    // FINAL REFINEMENTS FOR SOLUTION

    //I'll use beta again just to not allocating another useless variable
    int *idx_array;
    CHECK(hipMallocManaged(&idx_array, k*sizeof(int)));
    int count = 0;

    for (int i = 0; i < NSTREAM; ++i) {
        int ioffset = i * iElem;
        copy_arr<<<grid1St, block2, 0, stream[i]>>>(&alpha[ioffset], &beta[ioffset], m, ioffset);

        abs_array<<<grid1St, block2, 0, stream[i]>>>(beta,m, ioffset);
    }
    CHECK(hipDeviceSynchronize());

    float *sel_alpha;
    CHECK(hipMallocManaged(&sel_alpha, kSize));

    for (int i=0; i<m; i++){
        if(beta[i] <= 1/lambda){
            alpha[i] = 0;
        }
        else{
            idx_array[count] = i;
            sel_alpha[count] = alpha[i];
            count++;
        }
    }
   
    float *D1;
    float *D1_transp;
    float *D_transp;

    CHECK(hipMallocManaged(&D1, nSize*kSize));
    CHECK(hipMallocManaged(&D1_transp, kSize*nSize));
    CHECK(hipMallocManaged(&D_transp, mSize*mSize));

    gridShared.y = (n + blockShared.y - 1) / blockShared.y;
    gridShared.x = (m+blockShared.x - 1) / blockShared.x;
    transposeSmem<<<gridShared, blockShared>>>(D,D_transp,n,m); 
    CHECK(hipDeviceSynchronize());
    
    dim3 grid4((n + block.x - 1) / block.x, (k + block.y - 1) / block.y);

    subMatrix<<<grid4, block>>>(D1_transp,D_transp, idx_array,k,n);
    CHECK(hipDeviceSynchronize());
    
    gridShared.y = (k + blockShared.y - 1) / blockShared.y;
    gridShared.x = (n+blockShared.x - 1) / blockShared.x;
    transposeSmem<<<gridShared, blockShared>>>(D1_transp,D1, k,n);
    CHECK(hipDeviceSynchronize());

    // alpha(idx) = alpha(idx) - pinv(D1)*(D1*alpha(idx)-s);
    float *tmp_d1_alpha_mul;
    float *D1_pinv;
    float *tmp_pinvD1_par;
    CHECK(hipMallocManaged(&tmp_d1_alpha_mul, nSize));
    CHECK(hipMallocManaged(&D1_pinv, kSize*nSize));
    CHECK(hipMallocManaged(&tmp_pinvD1_par, kSize));
    
    for (int i = 0; i < NSTREAM; ++i) {
        int ioffset = i * iElem2;
        matrixMultStream<<<grid2MSt, blockShared, 0, stream[i]>>>(D1, sel_alpha, tmp_d1_alpha_mul, n,k,1,ioffset);
    }        
    
    CHECK(hipDeviceSynchronize()); 

    for (int i = 0; i < NSTREAM; ++i) {
            int ioffset = i * iElem2;
            matrixDiff<<<grid2St, block2, 0,stream[i]>>>(tmp_d1_alpha_mul, s,tmp_d1_alpha_mul,n, ioffset);
    }
    
    CHECK(hipDeviceSynchronize());
    
    float *mat_D1;
    CHECK(hipMallocManaged(&mat_D1,n*k));

    int iElem0 =  ((n*k)%NSTREAM == 0) ? (n*k)/NSTREAM : (n*k)/ NSTREAM+1;
    dim3 grid0MSt ((iElem0+blockShared2.x-1)/blockShared2.x);

    for (int i=0; i< NSTREAM; i++){
        int ioffset = i*iElem0;
        copy_matrix<<<grid0MSt, blockShared, 0,stream[i]>>>(D1,mat_D1,n,k, ioffset);
    }
  
    CHECK(hipDeviceSynchronize());
    
    moore_penrose_pinv(mat_D1, D1_pinv, n, k);
    CHECK(hipDeviceSynchronize());
    CHECK(hipFree(mat_D1));

    int iElem6 = k/NSTREAM+1;
    dim3 grid6St((iElem6+blockSize2-1)/blockSize2);
    
    int iElemM3 =  (k%NSTREAM == 0) ? k/NSTREAM : k/ NSTREAM+1;
    dim3 grid3MSt ((iElemM3/k+ blockShared.x - 1) / blockShared.x, (iElemM3 + blockShared.y - 1) / blockShared.y);

    for (int i=0; i< NSTREAM; i++){
        int ioffset = i*iElemM3;
        matrixMultStream<<<grid3MSt, blockShared, 0, stream[i]>>>(D1_pinv,tmp_d1_alpha_mul, tmp_pinvD1_par, k, n, 1, ioffset);
    } 
    
    CHECK(hipDeviceSynchronize());
    
    for (int i = 0; i < NSTREAM; ++i) {
        int ioffset = i * iElem6; 
        matrixDiff<<<grid6St, block2>>>(sel_alpha, tmp_pinvD1_par ,sel_alpha, k, ioffset);   
    }
    CHECK(hipDeviceSynchronize());


    for(int i = 0; i< k; i++){
         alpha[idx_array[i]] = sel_alpha[i];
    }

    for (int i = 0; i < NSTREAM; ++i)
        CHECK(hipStreamDestroy(stream[i]));

    CHECK(hipFree(idx_array));
    CHECK(hipFree(beta));
    CHECK(hipFree(alphaold));
    CHECK(hipFree(sel_alpha));
    CHECK(hipFree(D1));
    CHECK(hipFree(tmp_d1_alpha_mul));
    CHECK(hipFree(D1_pinv));
    CHECK(hipFree(tmp_pinvD1_par));
    CHECK(hipFree(D1_transp));
    CHECK(hipFree(D_transp));

    return;
}

int main(int argc, char *argv[]) {

    float *D, *Dinv, *s, *alpha;
    ulong nSize = N* sizeof(float);
    ulong mSize = M* sizeof(float);

    CHECK(hipSetDevice(0));

    CHECK(hipMallocManaged(&D, nSize*mSize));
    CHECK(hipMallocManaged(&Dinv, nSize*mSize));
    CHECK(hipMallocManaged(&s, nSize));
    CHECK(hipMallocManaged(&alpha, mSize));

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(transposeSmem), hipFuncCachePreferShared);


    hipEvent_t start, stop;
    float cuTime;

    hipEventCreate(&start);
    hipEventRecord(start,0);

    createDict_CPU(N,M,K,D,Dinv, s);
    k_limaps(N, M, K, s,D, Dinv, alpha);
    hipEventCreate(&stop);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cuTime, start,stop);
    printf("The resulting alpha is:\n");
    matrixDisplay(alpha, M,1);
    printf("\n\nTotal time in GPU: %f ms \n\n", cuTime);

    CHECK(hipFree(D));
    CHECK(hipFree(Dinv));
    CHECK(hipFree(s));
    CHECK(hipFree(alpha));

    hipError_t cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

    return 0;
	
}